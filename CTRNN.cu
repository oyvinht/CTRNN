#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "CTRNN.h"

__global__
void doUpdatePotentialsEulerCUDA(int netsize,
				 float stepsize,
				 float *activations,
				 float *biases,
				 float *externalCurrents,
				 float *potentials,
				 float *invTimeConstants,
				 float *weights)
{
  for (int to = 0; to < netsize; to++)
    {
      float input = externalCurrents[to];
      for (int from = 0; from < netsize; from++)
	{
	  input += weights[netsize * to + from] * activations[from];
	}
      potentials[to] += stepsize * invTimeConstants[to] * (input - potentials[to]);
    }
  for (int i = 0; i < netsize; i++)
    {
      activations[i] = 1 / ( 1 + exp(-(potentials[i] + biases[i])));
    }
}

namespace ctrnn
{
  
  struct CTRNN::impl
  {
    float *activations, *biases, *externalCurrents, *potentials;
    float *invTimeConstants;
    float *weights; // One segment of 'from'-weights per 'to' neuron
    int netsize;
    float stepsize;
    
  void init(int netsize, float stepsize)
    {
      this->netsize = netsize;
      this->stepsize = stepsize;

      hipMallocManaged(&activations, netsize * sizeof(float));
      hipMallocManaged(&biases, netsize * sizeof(float));
      hipMallocManaged(&externalCurrents, netsize * sizeof(float));
      hipMallocManaged(&invTimeConstants, netsize * sizeof(float));
      hipMallocManaged(&potentials, netsize * sizeof(float));
      hipMallocManaged(&weights, netsize * netsize * sizeof(float));
      /*
      activations = new float[netsize];
      biases = new float[netsize];
      externalCurrents = new float[netsize];
      invTimeConstants = new float[netsize];
      potentials = new float[netsize];
      weights = new float[netsize * netsize];
      */
      // Init properties of each neuron
      for (int i = 0; i < netsize; i++)
	{
	  biases[i] = 0.0f;
	  externalCurrents[i] = 0.0f;
	  invTimeConstants[i] = 1.0f;
	  potentials[i] = 0.0f;
	  activations[i] = 0.0f;//sigmoid(potentials[i] + biases[i]);
	}
      // Init weights (netsize * netsize)
      int from, to;
      for (to = 0; to < netsize; to++)
	{
	  for (from = 0; from < netsize; from++)
	    {
	      weights[netsize * to + from] = 0.0f;
	    }
	}
    }

   void updatePotentialsEulerCUDA()
    {
      doUpdatePotentialsEulerCUDA<<<1,1>>>(netsize,
				  stepsize,
				  activations,
				  biases,
				  externalCurrents,
				  potentials,
				  invTimeConstants,
				  weights);
    }
    
    void updatePotentialsRK4()
    {
      int from, to;
      float input;
      float *k1 = new float[netsize];
      float *k2 = new float[netsize];
      float *k3 = new float[netsize];
      float *k4 = new float[netsize];
      float *tmpAct = new float[netsize];
      float *tmpPot = new float[netsize];
      // Step 1
      for (to = 0; to < netsize; to++)
	{
	  input = externalCurrents[to];
	  for (from = 0; from < netsize; from++)
	    {
	      input += weights[netsize * to + from] * activations[from];
	    }
	  k1[to] = stepsize * invTimeConstants[to] * (input - potentials[to]);
	  tmpPot[to] = potentials[to] + (0.5 * k1[to]);
	  tmpAct[to] = sigmoid(tmpPot[to] + biases[to]);
	}
      // Step 2
      for (to = 0; to < netsize; to++)
	{
	  input = externalCurrents[to];
	  for (from = 0; from < netsize; from++)
	    {
	      input += weights[netsize * to + from] * tmpAct[from];
	    }
	  k2[to] = stepsize * invTimeConstants[to] * (input - tmpPot[to]);
	  tmpPot[to] = potentials[to] + (0.5 * k2[to]);
	}
      for (to = 0; to < netsize; to++)
	{
	  tmpAct[to] = sigmoid(tmpPot[to] + biases[to]);
	}
      // Step 3
      for (to = 0; to < netsize; to++)
	{
	  input = externalCurrents[to];
	  for (from = 0; from < netsize; from++)
	    {
	      input += weights[netsize * to + from] * tmpAct[from];
	    }
	  k3[to] = stepsize * invTimeConstants[to] * (input - tmpPot[to]);

	  tmpPot[to] = potentials[to] + k3[to];
	}
      for (to = 0; to < netsize; to++)
	{
	  tmpAct[to] = sigmoid(tmpPot[to] + biases[to]);
	}
      // Step 4
      for (to = 0; to < netsize; to++)
	{
	  input = externalCurrents[to];
	  for (from = 0; from < netsize; from++)
	    {
	      input += weights[netsize * to + from] * tmpAct[from];
	    }
	  k4[to] = stepsize * invTimeConstants[to] * (input - tmpPot[to]);
	  potentials[to] += (k1[to] + (2 * k2[to]) + (2 * k3[to]) + k4[to]) / 6;
	  activations[to] = sigmoid(potentials[to] + biases[to]);
	}
    }
  };
  CTRNN::CTRNN(int netsize, float stepsize) : pimpl{std::make_unique<impl>()}
  {
    pimpl->init(netsize, stepsize);
    return;
  }
  CTRNN::~CTRNN()
  {
    return;
  }
  float CTRNN::getActivation(int index)
  {
    hipDeviceSynchronize();
    return pimpl->activations[index];
  }
  float CTRNN::getBias(int index)
  {
    return pimpl->biases[index];
  }
  float CTRNN::getExternalCurrent(int index)
  {
    return pimpl->externalCurrents[index];
  }
  float CTRNN::getTimeConstant(int index)
  {
    return 1 / pimpl->invTimeConstants[index];
  }
  float CTRNN::getWeight(int fromIndex, int toIndex)
  {
    return pimpl->weights[pimpl->netsize * toIndex + fromIndex];
  }
  void CTRNN::setBias(int index, float bias)
  {
    pimpl->biases[index] = bias;
  }
  void CTRNN::setExternalCurrent(int index, float externalCurrent)
  {
    pimpl->externalCurrents[index] = externalCurrent;
  }
  void CTRNN::setTimeConstant(int index, float timeConstant)
  {
    pimpl->invTimeConstants[index] = 1 / timeConstant;
  }
  void CTRNN::setWeight(int fromIndex, int toIndex, float weight)
  {
    pimpl->weights[pimpl->netsize * toIndex + fromIndex] = weight;
  }
  void CTRNN::updatePotentials()
  {
    pimpl->updatePotentialsEulerCUDA();
    //pimpl->updatePotentialsRK4();
  }
}
